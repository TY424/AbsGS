#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  //the mean of the distance to the closest three points.
  torch::Tensor means = torch::full({P}, 0.0, float_opts);  //每个点都有一个最近距离

  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}